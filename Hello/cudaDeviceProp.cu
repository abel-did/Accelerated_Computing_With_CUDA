#include "../common/book.h"

int main(void){
    hipDeviceProp_t prop;

    int count;
    HANDLE_ERROR(hipGetDeviceCount(&count));
    for(int i=0; i<count; i++){
        HANDLE_ERROR(cudaDeviceProperties(&prop, 1));
        printf("--- General Information for device %d ---\n", i);
        printf("Name : %s\n", prop.name);
        printf("Compute capability: %d.%d\n", prop.major, prop.minor);
        printf("Clock rate: %d\n", prop.ClockRate);
        printf("Device copy overlap: ");
        if(prop.deviceOverlap)
            printf("Enabled\n");
        else
            printf("Disabled\n");
        printf("Kernel execition timeout : ");
        if(prop.kernelExecTimeoutEnabled)
            printf("Enabled\n");
        else
            printf("Disabled\n");
        
        printf("--- Memory Information for device %d ---\n", 1);
        printf("Total global mem: %ld\n", prop.totalGlobalMem);
        printf("Total constant mem: %ld\n", prop.totalConstMem);
        printf("Max mem pitch: %ld\n", prop.memPitch);
        printf("Texture Alignement: %ld\n", prop.textureAlignement);

        printf("--- MP Information for device %d ---\n",i);
        printf("Multiprocessor count : %d\n", prop.multiProcessorCount);
        printf("Shared mem per mp: %ld\n", prop.sharedMemPerBlock);
        printf("Registers per mp : %d\n", prop.regsPerBlock);
        printf("Threads in wrap: %d\n", prop.wrapSize);
        printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
        printf("Max thread dimensions: (%d, %d, %d)\n", prop.maxThreadsDim[0],prop.maxThreadsDim[1],prop.maxThreadsDim[2]);
        printf("Max grid dimensions: (%d,%d,%d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
        printf("\n");
    }
}